#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_BLOCKS 128
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK

hipEvent_t start, stop;

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if(hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s, %s\n", msg, hipGetErrorString(err));
		exit(-1);
	}
}

void printMat(unsigned size, float* mat){
    for(unsigned i=0; i<size; i++){
        printf("[");
        for(unsigned j=0; j<size-1; j++){
            printf("%.2f,\t", mat[i*size +j]);
        }
        printf("%.2f]\n", mat[i*size + size-1]);
    }
}


__global__
void vecAdditionKernel (int size, float *A, float *B, float *R) {
    int bi = threadIdx.x;
    int bj = threadIdx.y;
    int i  = blockIdx.x * blockDim.x + bi;
    int j  = blockIdx.y * blockDim.y + bj;

    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];

    float tmp = 0.0;
    for(unsigned t=0; t<size/TILE_WIDTH; t++){
        sharedA[bi][bj] = A[i*size + t*TILE_WIDTH + bj];
        sharedB[bi][bj] = B[(t*TILE_WIDTH+bi)*size + j];

        __syncthreads();//----------------------------------------

        for(unsigned k=0; k<TILE_WIDTH; k++) tmp += sharedA[bi][k] * sharedB[k][bj];

        __syncthreads();//----------------------------------------
    }
    R[i*size + j] = tmp;
}


int main( int argc, char** argv) {
    if(argc != 2){
        fprintf(stderr, "usage: ./dotProd size\n");
    }

    const unsigned size = atoi(argv[1]);
	const int bytes = size*size*sizeof(float);

    float** mat = (float***)malloc(3 * sizeof(float*));
    for(unsigned k=0; k<3; k++){
        mat[k] = (float*)malloc(size*size * sizeof(float));
    }
    float* A = mat[0];
    float* B = mat[1];
    float* R = mat[2];

    srand(1);
    for(unsigned i=0; i<size; i++){
        for(unsigned j=0; j<size; j++){
            float randf = (float)rand()/RAND_MAX*2.0 - 1.0; 
            A[i*size + j] = randf * 10.0;
            B[i*size + j] = 1.0;
            R[i*size + j] = 0.0;
        }
    }

	float *dA, *dB, *dR;
	hipMalloc((void**) &dA, bytes);
	hipMalloc((void**) &dB, bytes);
	hipMalloc((void**) &dR, bytes);
	checkCUDAError("mem allocation");

    hipEvent_t start_kernel, end_kernel, start_send, end_send, start_recv, end_recv;
    hipEventCreate(&start_kernel   );
    hipEventCreate(&end_kernel     );
    hipEventCreate(&start_send     );
    hipEventCreate(&end_send       );
    hipEventCreate(&start_recv     );
    hipEventCreate(&end_recv       );

	hipEventRecord(start_send);//>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
	hipMemcpy(da, a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(db, b, bytes, hipMemcpyHostToDevice);
	checkCUDAError("memcpy h->d");
	hipEventRecord(end_send);//<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	hipEventSynchronize(end_send);

	hipEventRecord(start_kernel);//>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
	dotProdKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>>(size, da, db, dc);
	checkCUDAError("kernel invocation");
	hipEventRecord(end_kernel);//<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	hipEventSynchronize(end_kernel);

	hipEventRecord(start_recv);//>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
	hipMemcpy(c, dc, bytes, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");
	hipEventRecord(end_recv);//<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	hipEventSynchronize(end_recv);


	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_send, end_send);
    printf("memcpy h->d time: %f", milliseconds);
	hipEventElapsedTime(&milliseconds, start_recv, end_recv);
    printf("memcpy d->h time: %f", milliseconds);
	hipEventElapsedTime(&milliseconds, start_kernel, end_kernel);
    printf("kernel time: %f", milliseconds);




	// print matrix
#ifdef DEBUG
    //printf("A:\n"); printMat(size, A);
    //printf("B:\n"); printMat(size, B);
    printf("A*B:\n"); printMat(size, R);
    for(unsigned i=0; i<size; i++)
        for(unsigned j=0; j<size; j++)
            R[i][j] = 0.0;
    dotProd(size, B, A, R);
    printf("B*A:\n"); printMat(size, R);
#endif

	hipFree(da); hipFree(db); hipFree(dc);
	checkCUDAError("mem free");
    free(A); free(B); free(R); free(mat);
    return(0);
}
